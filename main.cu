#include <iostream>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define WIDTH 64

using crngState = hiprandStatePhilox4_32_10_t;

/* Each thread gets same seed, a different sequence
   number, no offset */
__global__ void setup_curand(crngState *state, unsigned long seed, unsigned dim) {
    uint32_t xid = blockIdx.z * blockDim.x + threadIdx.x;
    uint32_t id  = blockIdx.y*dim + xid;
    if(xid >= dim) return;

    //printf("blocks: %d %d %d / %d %d %d\n", blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
    //printf("threads: %d %d %d / %d %d %d\n", threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z);
    hiprand_init(seed, id, 0, &state[id]);
}

__global__ void gen_image_kernel(crngState *state, unsigned int *result, unsigned dim) {
    uint32_t xid = blockIdx.z * blockDim.x + threadIdx.x;
    uint32_t id  = blockIdx.y*dim + xid;
    if(xid >= dim) return;

    crngState localState = state[id];
    unsigned int x = hiprand(&localState);
    state[id] = localState;
    result[id] = x;
}

unsigned int *gen_image_gpu(unsigned dim, unsigned int seed) {
    unsigned int *image = (unsigned int *)malloc(dim*dim*sizeof(unsigned int));
    unsigned int *image_d;
    crngState *state;
    
    hipMalloc(&image_d, dim*dim*sizeof(unsigned int));
    dim3 dims(1, dim, (dim+WIDTH-1)/WIDTH);
    
    hipMalloc(&state, WIDTH*sizeof(crngState));
    setup_curand<<< dims, WIDTH >>>(state, seed, dim);
    gen_image_kernel<<< dims, WIDTH >>>(state, image_d, dim);
    hipMemcpy(image, image_d, dim*dim*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipFree(state);
    hipFree(image_d);

    return image;
}

int main(int argc, char *argv[]) {
    if(argc != 3) {
        std::cout << "Usage: " << argv[0] << " <dim> <steps>\n";
        return 1;
    }
    int dim = atoi(argv[1]);
    int steps = atoi(argv[2]);

    unsigned int *img = gen_image_gpu(dim, steps);

    for(int i=0; i<dim; i += (dim+9)/10) {
        uint32_t id = i*dim + i;
        std::cout << i << ": " << img[id] << " " << img[id+1] << std::endl;
    }

    free(img);
    return 0;
}
